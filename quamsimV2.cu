/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iterator>
#include <iomanip>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
using namespace std;


/**
 * CiiUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorQuantumComputing(float *qbit_input_quantum_state, float *qbit_quantum_gate, float *qbit_output_quantum_state,int qbit_value,  int numElements)
{
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;

        if(i < numElements && i % int (__powf(2,(qbit_value+1)))<int (__powf(2,(qbit_value))))
        {
            qbit_output_quantum_state[i] = (qbit_quantum_gate[0] * qbit_input_quantum_state[i] + qbit_quantum_gate[1] * qbit_input_quantum_state[i | (1 << qbit_value)]);
            qbit_output_quantum_state[i | (1 << qbit_value)] = (qbit_quantum_gate[2] * qbit_input_quantum_state[i] + qbit_quantum_gate[3] * qbit_input_quantum_state[i | (1 << qbit_value)]);

        }
    
}

/**
 * Host main routine
 */
int
main(void)
{





    FILE * FP;
    FP=fopen("input.txt","r");
    int number_of_lines;
    char element;
    float qbit_quantum_gate_temp[2][2];


    if(FP==NULL){
        cout<<"File not found"<<endl;
        return 0;
    }

    while (EOF != (element=getc(FP))) {
        if ('\n' == element)
            number_of_lines=number_of_lines+1;
    }
    int numElements = number_of_lines-4;


      float* qbit_input_quantum_state = new float [(number_of_lines-3)];
    float* qbit_output_quantum_state = new float [(number_of_lines-4)];

    hipMallocManaged(&qbit_input_quantum_state, number_of_lines-3*sizeof(float));
    hipMallocManaged(&qbit_output_quantum_state, numElements*sizeof(float));


    FP=fopen("input.txt","r");

    int i=0;
    while(fscanf(FP, "%f %f", &qbit_quantum_gate_temp[i][0], &qbit_quantum_gate_temp[i][1]) != EOF)
    {
        i++;
        if (i>1)
        {
            i = 0;
            while (fscanf(FP, "%f ", &qbit_input_quantum_state[i]) != EOF)
            {
                i++;
            }
            break;
        }
    }

    int qbit_value=qbit_input_quantum_state[numElements];



    float* qbit_quantum_gate=new float [4];
    hipMallocManaged(&qbit_quantum_gate, 4*sizeof(float));
    qbit_quantum_gate[0]=qbit_quantum_gate_temp[0][0];
    qbit_quantum_gate[1]=qbit_quantum_gate_temp[0][1];
    qbit_quantum_gate[2]=qbit_quantum_gate_temp[1][0];
    qbit_quantum_gate[3]=qbit_quantum_gate_temp[1][1];






    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size







    // Verify that allocations succeeded
    if (qbit_input_quantum_state == NULL || qbit_quantum_gate == NULL || qbit_output_quantum_state == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

   // cudaMallocManaged(&qbit_input_quantum_state, number_of_lines-3*sizeof(float));
    //cudaMallocManaged(&qbit_output_quantum_state, numElements*sizeof(float));
   // cudaMallocManaged(&qbit_quantum_gate, 4*sizeof(float));

    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorQuantumComputing<<<blocksPerGrid, threadsPerBlock>>>(qbit_input_quantum_state, qbit_quantum_gate, qbit_output_quantum_state,qbit_value, numElements);
    hipDeviceSynchronize();
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    for(int k=0;k<number_of_lines-4;k++)
    {
        cout<<fixed<<setprecision(3)<<qbit_output_quantum_state[k]<<endl;
    }

    // Free host memory
  //  free(qbit_input_quantum_state);
   // free(qbit_quantum_gate);
  //  free(qbit_output_quantum_state);


    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
 //   err = cudaDeviceReset();

   // if (err != cudaSuccess)
   // {
     //   fprintf(stderr, "Failed to deinitialize the device! error=%s\n", cudaGetErrorString(err));
       // exit(EXIT_FAILURE);
   // }

    return 0;
}

